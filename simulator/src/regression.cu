#include "hip/hip_runtime.h"
#include <hip/hip_runtime_api.h>
#include <iostream>
#include <math.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "regression.h"

#define xzt(x,z,t) ((x) + (z)*(X) + ((t)%4)*(X)*(Z))
#define xzt2(x,z,t) ((x) + (z)*(X) + ((t)%4)*(X)*(Z))
#define inbounds(x,z,offset) (((x)>=(offset) && (z)>=(offset) && (x)<(X)-(offset) && (z)<(Z)-(offset)))

//precisam ser iguais
#define WARP_SIZE (32)
#define BLOCK_SIZE (32)

#define prec_deriv (4)
#include "deriv_macros.h"


const dim3 threadGrid(BLOCK_SIZE, BLOCK_SIZE);

float *P, *cquad, *source, *record_buffer, *initial, *recording_h;
float *P_ub, *P_uf, *grad, *observed, *adj_source, *grad_h, *simulated_h, *adj_source_h;
float *P_uf_full;
int *pos_source_x, *pos_source_z, *pos_sensor_x, *pos_sensor_z;
int X, Z, T, n_source, n_sensor;
int allocated = 0;
unsigned int n_blocksX, n_blocksZ, n_blocksS, n_blocksF;
dim3 blockGrid;


__global__ void
simulateFrame (float* P, float *cquad, int X, int Z, int t)
{
    //coordenadas no bloco
    const int x = threadIdx.x;
    const int z = threadIdx.y;

    //coordenadas em Ps (shared)
    const int x_s = threadIdx.x + prec_deriv;
    const int z_s = threadIdx.y + prec_deriv;

    //coordenadas da origem do bloco
    const int x_b = blockIdx.x * blockDim.x;
    const int z_b = blockIdx.y * blockDim.y;

    //coordenadas em P (global)
    const int x_g = x_b + x;
    const int z_g = z_b + z;

    const int shared_width = BLOCK_SIZE + 2*prec_deriv;
    const int tam_shared = (shared_width)*(shared_width);

    //nao vale a pena colocar cquad e P(t-2) na memoria shared de acordo com os testes
    __shared__ float Ps[shared_width][shared_width];


    if(z==0) //first warp in block
    {
	//copy P to shared memory
	for(int id=x; id<tam_shared; id+=WARP_SIZE)
	{
	    //coordenada dentro de Ps (shared) sendo lida de P
	    const int xx = id/shared_width;
	    const int zz = id%shared_width;

	    //coordenada correspondente na memoria global
	    const int x_c = x_b - prec_deriv + xx;
	    const int z_c = z_b - prec_deriv + zz;

	    //retirar if adicionando o anel de zeros
	    if(inbounds(x_c, z_c, prec_deriv))
		Ps[zz][xx] = P[xzt(x_c, z_c, t-1)];
	    else
		Ps[zz][xx] = 0.0f;
	}
    }
    __syncthreads();
    

    float lap = deriv_x(Ps, x_s, z_s) + deriv_z(Ps, x_s, z_s);

    //assume que nao tem anel de zeros forcados por indice, entao usa as bordas como anel
    if(inbounds(x_g, z_g, prec_deriv))
	P[xzt(x_g, z_g, t)] = -P[xzt(x_g, z_g, t - 2)] + 2 * Ps[z_s][x_s] + cquad[xzt (x_g, z_g, 0)] * lap;
    else
	P[xzt(x_g,z_g,t)] = 0.0f;
}


__global__ void
somaFonte(float *P, int X, int Z, int T, int t, int *pos_source_x, int *pos_source_z, float *source, int n_source, int flip, int idx=-1)
{
    const int indexF = threadIdx.x + blockDim.x*threadIdx.y + blockIdx.x*(BLOCK_SIZE*BLOCK_SIZE);
    const int n = indexF; 

    if(n>=n_source || (idx!=-1 && idx!=n))
	return;

    float fonte;
    if(flip)
	fonte = source[n * T + T - 1 - t];
    else
	fonte = source[n*T + t];

    P[xzt(pos_source_x[n], pos_source_z[n], t)] += fonte;
}


__global__ void
gravaBufferSensores2ordem(float *P, float *recording, int X, int Z, int T, int t, int *pos_sensor_x, int *pos_sensor_z, int n_sensor)
{
    //coordenadas no bloco
    const int indexS = threadIdx.x + blockDim.x*threadIdx.y + blockIdx.x*(BLOCK_SIZE*BLOCK_SIZE);
    const int n = indexS; 
    if(n<n_sensor)
	recording[n*T + t] = P[xzt2(pos_sensor_x[n], pos_sensor_z[n], t)];
}


void
allocate_mem_simulate()
{
    hipMalloc(&P, X * Z * 4 * sizeof (float));	//pressao direta
    hipMalloc(&initial, X * Z * 2 * sizeof (float));	//pressao direta
    hipMalloc(&cquad, X * Z * sizeof (float));	//campo de velocidades
    hipMalloc(&source, T * n_source * sizeof (float));	//termos de fonte
    hipMalloc(&record_buffer, T * n_sensor * sizeof (float));	//buffer dos sensores

    hipMalloc(&pos_source_x, n_source*sizeof(int)); //posicoes das fontes
    hipMalloc(&pos_source_z, n_source*sizeof(int)); //posicoes das fontes
    hipMalloc(&pos_sensor_x, n_sensor*sizeof(int)); //posicoes dos sensores
    hipMalloc(&pos_sensor_z, n_sensor*sizeof(int)); //posicoes dos sensores 

    hipHostMalloc(&recording_h, T*n_sensor*sizeof(float));
    //recording_h = (float*)malloc(T*n_sensor*sizeof(float));

    allocated = 1;
}


void
free_mem_simulate()
{
    hipFree (P);
    hipFree (initial);
    hipFree (cquad);
    hipFree (source);
    hipFree (record_buffer);

    hipFree(pos_source_x);
    hipFree(pos_source_z);
    hipFree(pos_sensor_x);
    hipFree(pos_sensor_z);

    hipFree(recording_h);

    allocated = 0;
}


extern "C" void
init_memory_sim(int x, int z, int t, float *cq, 
	int ns, int *ps_x, int *ps_z, int nm, int *pm_x, int *pm_z, 
	float *src, float *init, float **rec)
{
    X = x; 
    Z = z; 
    T = t; 
    n_source = ns; 
    n_sensor = nm;

    if(allocated)
	free_mem_simulate();

    hipDeviceReset();

    allocate_mem_simulate();

    n_blocksX = X/BLOCK_SIZE;
    n_blocksZ = Z/BLOCK_SIZE;
    n_blocksS = n_sensor/(BLOCK_SIZE*BLOCK_SIZE);
    n_blocksF = n_source/(BLOCK_SIZE*BLOCK_SIZE);

    if(n_blocksX*BLOCK_SIZE != X)
	n_blocksX++;
    if(n_blocksZ*BLOCK_SIZE != Z)
	n_blocksZ++;
    if(n_blocksS*(BLOCK_SIZE*BLOCK_SIZE) != n_sensor) 
	n_blocksS++;
    if(n_blocksF*(BLOCK_SIZE*BLOCK_SIZE) != n_source) 
	n_blocksF++;


    blockGrid = {n_blocksX, n_blocksZ, 1};


    *rec = recording_h;
    hipMalloc(&P, X * Z * 4 * sizeof (float));	//pressao direta
    hipMemcpy(cquad, cq, X * Z * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(source, src, T * n_source * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_x, pm_x, n_sensor*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_z, pm_z, n_sensor*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_source_x, ps_x, n_source*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_source_z, ps_z, n_source*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(initial, init, X*Z*2*sizeof(float), hipMemcpyHostToDevice);
}


extern "C" void
setCquad(float *cq)
{
    hipMemcpy(cquad, cq, X * Z * sizeof (float), hipMemcpyHostToDevice);
}


extern "C" void
set_source(int ns, int *sx, int *sz, float *src)
{
    n_source = ns;
    hipMemcpy(pos_sensor_x, sx, ns * sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_z, sz, ns * sizeof (int), hipMemcpyHostToDevice);
    hipMemcpy(source, src, ns * T * sizeof (float), hipMemcpyHostToDevice);
}


extern "C" void
cuda_simulate2 (int en_out, int idx_source)
{
    FILE *pipeout;
    float *frame_buffer;
    if (en_out) 
    {
	char mpegCom[500];
	sprintf(mpegCom, "ffmpeg -y -f rawvideo -vcodec rawvideo -pix_fmt gray -s %ix%i -r 20 -i - -f mp4 -q:v 5 -an -vcodec h264 -crf 0 output/output1P.mp4 -nostats -loglevel quiet", X, Z);    
	pipeout = popen(mpegCom, "w");    
	hipHostMalloc(&frame_buffer, X*Z*sizeof(float));
    }

    //copia condicoes iniciais
    hipMemset(P, 0, X*Z*4*sizeof(float));
    hipMemcpy(P, initial, X*Z*2*sizeof(float), hipMemcpyDeviceToDevice);

    for (int t = 0; t < T; t++)
    {
	//primeiros 2 frames sao condicao de contorno, logo nao calculados
	if (t > 1)
	{
	    simulateFrame <<<blockGrid, threadGrid>>> (P, cquad, X, Z, t);
	    hipDeviceSynchronize ();

	    somaFonte<<<n_blocksF, threadGrid>>>(P, X, Z, T, t, pos_source_x, pos_source_z, source, n_source, 0, idx_source);
	    hipDeviceSynchronize ();
	}
		
	// grava resultado nos sensores
	//gravaBufferSensores2ordem<<<1, n_sensor>>>(P, record_buffer, X, Z, T, t, pos_sensor_x, pos_sensor_z);
	gravaBufferSensores2ordem<<<n_blocksS, threadGrid>>>(P, record_buffer, X, Z, T, t, pos_sensor_x, pos_sensor_z, n_sensor);
	hipDeviceSynchronize();

	if (en_out)
	{
	    hipMemcpy(frame_buffer, &P[xzt(0,0,t)], X*Z*sizeof(float), hipMemcpyDeviceToHost);
	    writeFramePipe (pipeout, frame_buffer, X, Z, t, pos_sensor_x, pos_sensor_z, n_sensor);
	}
    }

    if(en_out)
    {
	fflush(pipeout);
	pclose(pipeout);
	hipFree(frame_buffer);
    }
    
    hipMemcpy (recording_h, record_buffer, T * n_sensor * sizeof (float), hipMemcpyDeviceToHost);
}

