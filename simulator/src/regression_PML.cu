#include "hip/hip_runtime.h"
#include <iostream>
#include <math.h>
#include <ctype.h>
#include <stdio.h>
#include <stdlib.h>
#include <unistd.h>
#include "regression.h"

#define xzt(x,z,t) ((x) + (z)*(X) + ((t)%2)*(X)*(Z))
#define inbounds(x,z,offset) (((x)>=(offset) && (z)>=(offset) && (x)<(X)-(offset) && (z)<(Z)-(offset)))

//precisam ser iguais
#define WARP_SIZE (32)
#define BLOCK_SIZE (32)

#define prec_deriv (4)
#include "deriv_macros.h"

typedef void (*adj_func)(float*, float*);

float *Ax, *Az, *Px, *Pz, *P, *d_x, *d_z, *dx, *dz, *dt, *cquad, *source, *record_buffer;
float *integral_source, *recording_h, *initial;
float *Ax_f, *Az_f, *Px_f, *Pz_f, *Ax_b, *Az_b, *Px_b, *Pz_b;
float *P_ub, *P_uf, *grad, *observed, *observed_h, *adj_source, *grad_h, *simulated_h;
float *adj_source_h, *integral_adjsource, *int_revsource;
int *pos_sensor_x, *pos_sensor_z, *pos_source_x, *pos_source_z; 
int *pos_revert_x, *pos_revert_z, n_revert;
float *rec_revert;
int X, Z, T, n_source, n_sensor;
int allocated = 0, allocated_reg = 0;
unsigned int n_blocksX, n_blocksZ, n_blocksS, n_blocksF, n_blocksR;

dim3 blockGrid;
const dim3 threadGrid(BLOCK_SIZE, BLOCK_SIZE);


void
allocate_mem_simulate()
{
    // aloca memória no device
    hipMalloc(&Px, X*Z*2*sizeof(float)); //pressao direta x
    hipMalloc(&Pz, X*Z*2*sizeof(float)); //pressao direta z
    hipMalloc(&Ax, X*Z*2*sizeof(float)); //velocidade x
    hipMalloc(&Az, X*Z*2*sizeof(float)); //velocidade z
    hipMalloc(&dx, sizeof(float)); //discretização em x
    hipMalloc(&dz, sizeof(float)); //discretização em z
    hipMalloc(&dt, sizeof(float)); //discretização em t
    hipMalloc(&P, X*Z*sizeof(float)); //pressao direta
    hipMalloc(&d_x, X*Z*sizeof(float)); //atenuação x
    hipMalloc(&d_z, X*Z*sizeof(float)); //atenuação x
    hipMalloc(&cquad, X*Z*sizeof(float)); //campo de velocidades meio
    hipMalloc(&source, T*n_source*sizeof(float)); //termos de fonte
    hipMalloc(&pos_source_x, n_source*sizeof(int)); //posicoes das fontes
    hipMalloc(&pos_source_z, n_source*sizeof(int)); //posicoes das fontes
    hipMalloc(&pos_sensor_x, n_sensor*sizeof(int)); //posicoes dos sensores
    hipMalloc(&pos_sensor_z, n_sensor*sizeof(int)); //posicoes dos sensores 
    hipMalloc(&record_buffer, T*n_sensor*sizeof(float)); //buffer dos sensores
    hipMalloc(&integral_source, n_source*sizeof(float)); //termos de fonte

    hipHostMalloc(&recording_h, T*n_sensor*sizeof(float));

    allocated = 1;
}

void
free_mem_simulate()
{
    hipFree(pos_sensor_x);
    hipFree(pos_sensor_z);
    hipFree(pos_source_x);
    hipFree(pos_source_z);
    hipFree(integral_source);
    hipFree(dx);
    hipFree(dz);
    hipFree(dt);
    hipFree(P);
    hipFree(Px);
    hipFree(Pz);
    hipFree(Ax);
    hipFree(Az);
    hipFree(d_x);
    hipFree(d_z);
    hipFree(record_buffer);
    hipFree(cquad);
    hipFree(source);

    hipFree(recording_h);

    allocated = 0;
}


extern "C" void
init_memory_sim(int x, int z, int t, float *cq, float *const_vec, float *d_x_h, float *d_z_h, 
	int ns, int *ps_x, int *ps_z, int nm, int *pm_x, int *pm_z, int ppe,
	float *src, float *init, float **rec)
{
    X = x; 
    Z = z; 
    T = t; 
    n_source = ns; 
    n_sensor = nm;

    if(allocated)
	free_mem_simulate();

    hipDeviceReset();

    allocate_mem_simulate();

    n_blocksX = X/BLOCK_SIZE;
    n_blocksZ = Z/BLOCK_SIZE;
    n_blocksS = n_sensor/(BLOCK_SIZE*BLOCK_SIZE);
    n_blocksF = n_source/(BLOCK_SIZE*BLOCK_SIZE);

    if(n_blocksX*BLOCK_SIZE != X)
	n_blocksX++;
    if(n_blocksZ*BLOCK_SIZE != Z)
	n_blocksZ++;
    if(n_blocksS*(BLOCK_SIZE*BLOCK_SIZE) != n_sensor) 
	n_blocksS++;
    if(n_blocksF*(BLOCK_SIZE*BLOCK_SIZE) != n_source) 
	n_blocksF++;


    blockGrid = {n_blocksX, n_blocksZ, 1};


    *rec = recording_h;
    hipMalloc(&P, X * Z * 2 * sizeof (float));	//pressao direta
    hipMemcpy(cquad, cq, X * Z * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(source, src, T * n_source * sizeof (float), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_x, pm_x, n_sensor*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_sensor_z, pm_z, n_sensor*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_source_x, ps_x, n_source*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_source_z, ps_z, n_source*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(initial, init, X*Z*2*sizeof(float), hipMemcpyHostToDevice);

    hipMemcpy(dx, const_vec, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dz, const_vec+1, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(dt, const_vec+2, sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_x, d_x_h, X*Z*sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_z, d_z_h, X*Z*sizeof(float), hipMemcpyHostToDevice);
}


extern "C" void
setCquad(float *cq)
{
    hipMemcpy(cquad, cq, X * Z * sizeof (float), hipMemcpyHostToDevice);
}


extern "C" void
setSource(int ns, int *ps_x, int *ps_z)
{
    n_source = ns;

    hipMemcpy(pos_source_x, ps_x, n_source*sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(pos_source_z, ps_z, n_source*sizeof(int), hipMemcpyHostToDevice);
}


__global__ void
setRec(float *Px, float *Pz, int X, int Z, int T, int t, int n_revert, int *pos_revert_x, int *pos_revert_z, float *revert)
{
    const int indexF = threadIdx.x + blockDim.x*threadIdx.y + blockIdx.x*(BLOCK_SIZE*BLOCK_SIZE);
    const int n = indexF; 

    if(n>=n_revert)
	return;

    Px[xzt(pos_revert_x[n], pos_revert_z[n], t)] = revert[n*T + T-1-t]/2;
    Pz[xzt(pos_revert_x[n], pos_revert_z[n], t)] = revert[n*T + T-1-t]/2;
    //Px[xzt(pos_revert_x[n], pos_revert_z[n], t)] = 0;
    //Pz[xzt(pos_revert_x[n], pos_revert_z[n], t)] = 0;
}


__global__ void
somaFonteIntegral(float *Px, float *Pz, int X, int Z, int T, int t, int n_fonte, int *pos_source_x, int *pos_source_z, float *source, float *integral, int flip, int idx=-1)
{
    const int indexF = threadIdx.x + blockDim.x*threadIdx.y + blockIdx.x*(BLOCK_SIZE*BLOCK_SIZE);
    const int n = indexF; 

    if(n>=n_fonte || (idx!=-1 && idx!=n))
	return;

    if(flip)
	integral[n] += source[n * T + T - 1 - t];
    else
	integral[n] += source[n*T + t];

    Px[xzt(pos_source_x[n], pos_source_z[n], t)] += integral[n]/2;
    Pz[xzt(pos_source_x[n], pos_source_z[n], t)] += integral[n]/2;
}


__global__ void
gravaBufferSensores(float *Px, float *Pz, float *buffer, int X, int Z, int T, int t, int n_sensors, int *pos_sensor_x, int *pos_sensor_z)
{
    const int indexS = threadIdx.x + blockDim.x*threadIdx.y + blockIdx.x*(BLOCK_SIZE*BLOCK_SIZE);
    const int n = indexS; 


    if(n<n_sensors)
    {
	buffer[n*T + t] = Px[xzt(pos_sensor_x[n], pos_sensor_z[n], t)] + Pz[xzt(pos_sensor_x[n], pos_sensor_z[n], t)];
    }
    //else
//	printf("index: %d\n", n);

}


__global__ void
somaFrames(float *destino, float *origem1, float *origem2, int X, int Z)
{
    const int x = threadIdx.x;
    const int z = threadIdx.y;

    //coordenadas da origem do bloco
    const int x_b = blockIdx.x * blockDim.x;
    const int z_b = blockIdx.y * blockDim.y;

    //coordenadas em P (global)
    const int x_g = x_b + x;
    const int z_g = z_b + z;

    if(inbounds(x_g, z_g, prec_deriv))
    {
        destino[xzt(x_g,z_g,0)] = origem1[xzt(x_g,z_g,0)] + origem2[xzt(x_g,z_g,0)];
    }
    else
    {
        return;
    }
}


__global__
void simulateFrameP(float *Px, float *Pz, float *Ax, float *Az,
			float *d_x, float *d_z, float *cquad, float *dx, float *dz, float *dt,
			int t, int X, int Z, int revert=0)
{
    //coordenadas no bloco
    const int x = threadIdx.x;
    const int z = threadIdx.y;

    //coordenadas em Ps (shared)
    const int x_s = threadIdx.x + prec_deriv;
    const int z_s = threadIdx.y + prec_deriv;

    //coordenadas da origem do bloco
    const int x_b = blockIdx.x * blockDim.x;
    const int z_b = blockIdx.y * blockDim.y;

    //coordenadas em P (global)
    const int x_g = x_b + x;
    const int z_g = z_b + z;

    const int shared_width = BLOCK_SIZE + 2*prec_deriv;
    const int tam_shared = (shared_width)*(shared_width);

    //nao vale a pena colocar cquad e P(t-2) na memoria shared de acordo com os testes
    __shared__ float Asx[shared_width][shared_width];
    __shared__ float Asz[shared_width][shared_width];

    if(z==0) //first warp in block
    {
	//copy P to shared memory
	for(int id=x; id<tam_shared; id+=WARP_SIZE)
	{
	    //coordenada dentro de Ps (shared) sendo lida de P
	    const int xx = id/shared_width;
	    const int zz = id%shared_width;

	    //coordenada correspondente na memoria global
	    const int x_c = x_b - prec_deriv + xx;
	    const int z_c = z_b - prec_deriv + zz;

	    //retirar if adicionando o anel de zeros
	    if(inbounds(x_c, z_c, prec_deriv))
	    {
		Asx[zz][xx] = Ax[xzt(x_c, z_c, t-1)];
		Asz[zz][xx] = Az[xzt(x_c, z_c, t-1)];
	    }
	    else
	    {
		Asx[zz][xx] = 0.0f;
		Asz[zz][xx] = 0.0f;
	    }
	}
    }
    __syncthreads();
    
    const float dAxdx = derivPMLdx(Asx, 1, x_s, z_s);
    const float dAzdz = derivPMLdz(Asz, 1, x_s, z_s);
    const float d_xx = (d_x[xzt(x_g+1,z_g,0)]);
    const float d_zz = (d_z[xzt(x_g,z_g+1,0)]);

    if(inbounds(x_g, z_g, prec_deriv))
    {
	if(!revert)
	{
	    Px[xzt(x_g,z_g,t)] = Px[xzt(x_g,z_g,t-1)] * (1 - d_xx) + cquad[xzt(x_g,z_g,0)] * *dt/(*dx) * dAxdx;
	    Pz[xzt(x_g,z_g,t)] = Pz[xzt(x_g,z_g,t-1)] * (1 - d_zz) + cquad[xzt(x_g,z_g,0)] * *dt/(*dx) * dAzdz;
	}
	else
	{
	    Px[xzt(x_g,z_g,t)] = (Px[xzt(x_g,z_g,t-1)] - cquad[xzt(x_g,z_g,0)] * (*dt/(*dx)) * dAxdx);// /(1 - d_xx);
	    Pz[xzt(x_g,z_g,t)] = (Pz[xzt(x_g,z_g,t-1)] - cquad[xzt(x_g,z_g,0)] * (*dt/(*dx)) * dAzdz);// /(1 - d_zz);
	}
    }
    else
    {
	//Px[xzt(x_g,z_g,t)] = 0.0f;
	//Pz[xzt(x_g,z_g,t)] = 0.0f;
    }


}


__global__
void simulateFrameA(float *Px, float *Pz, float *Ax, float *Az,
		    	float *d_x, float *d_z, float *cquad, float *dx, float *dz, float *dt,
			int t, int X, int Z, int revert=0)
{
    //coordenadas no bloco
    const int x = threadIdx.x;
    const int z = threadIdx.y;

    //coordenadas em Ps (shared)
    const int x_s = threadIdx.x + prec_deriv;
    const int z_s = threadIdx.y + prec_deriv;

    //coordenadas da origem do bloco
    const int x_b = blockIdx.x * blockDim.x;
    const int z_b = blockIdx.y * blockDim.y;

    //coordenadas em P (global)
    const int x_g = x_b + x;
    const int z_g = z_b + z;

    const int shared_width = BLOCK_SIZE + 2*prec_deriv;
    const int tam_shared = (shared_width)*(shared_width);

    //nao vale a pena colocar cquad e P(t-2) na memoria shared de acordo com os testes
    __shared__ float Psx[shared_width][shared_width];
    __shared__ float Psz[shared_width][shared_width];

    if(z==0) //first warp in block
    {
	//copy P to shared memory
	for(int id=x; id<tam_shared; id+=WARP_SIZE)
	{
	    //coordenada dentro de Ps (shared) sendo lida de P
	    const int xx = id/shared_width;
	    const int zz = id%shared_width;

	    //coordenada correspondente na memoria global
	    const int x_c = x_b - prec_deriv + xx;
	    const int z_c = z_b - prec_deriv + zz;

	    //retirar if adicionando o anel de zeros
	    if(inbounds(x_c, z_c, prec_deriv))
	    {
		Psx[zz][xx] = Px[xzt(x_c, z_c, t)];
		Psz[zz][xx] = Pz[xzt(x_c, z_c, t)];
	    }
	    else
	    {
		Psx[zz][xx] = 0.0f;
		Psz[zz][xx] = 0.0f;
	    }
	}
    }
    __syncthreads();
    
    const float dPxdx = derivPMLdx(Psx, 0, x_s, z_s);
    const float dPxdz = derivPMLdz(Psx, 0, x_s, z_s);
    const float dPzdx = derivPMLdx(Psz, 0, x_s, z_s);
    const float dPzdz = derivPMLdz(Psz, 0, x_s, z_s);

    const float d_xx = (d_x[xzt(x_g+1,z_g,0)]);
    const float d_zz = (d_z[xzt(x_g,z_g+1,0)]);


    //assume que nao tem anel de zeros forcados por indice, entao usa as bordas como anel
    if(inbounds(x_g, z_g, prec_deriv))
    {
	if(!revert)
	{
	    Ax[xzt(x_g,z_g,t)] = Ax[xzt(x_g,z_g,t-1)] * (1 - d_xx) + *dt/(*dx) * (dPxdx + dPzdx);
	    Az[xzt(x_g,z_g,t)] = Az[xzt(x_g,z_g,t-1)] * (1 - d_zz) + *dt/(*dx) * (dPxdz + dPzdz);
	}
	else
	{
	    Ax[xzt(x_g,z_g,t)] = (Ax[xzt(x_g,z_g,t-1)] - *dt/(*dx) * (dPxdx + dPzdx));// /(1 - d_xx);
	    Az[xzt(x_g,z_g,t)] = (Az[xzt(x_g,z_g,t-1)] - *dt/(*dx) * (dPxdz + dPzdz));// /(1 - d_zz);
	}
    }
    else
    {
	//Ax[xzt(x_g,z_g,t)] = 0.0f;
        //Az[xzt(x_g,z_g,t)] = 0.0f;
    }
}


extern "C"
void cuda_simulate(int en_out, int idx_source)
{
    FILE *pipeout;
    float *frame_buffer;
    if(en_out)
    {
	char mpegCom[500];
	sprintf(mpegCom, "ffmpeg -y -f rawvideo -vcodec rawvideo -pix_fmt gray -s %ix%i -r 20 -i - -f mp4 -q:v 5 -an -vcodec h264 -crf 0 output/outputPML.mp4 -nostats -loglevel quiet", X, Z);
	pipeout = popen(mpegCom, "w");    
	hipHostMalloc(&frame_buffer, X*Z*sizeof(float));
    }



    // limpa diretorios de saída
    int saida = 0;
    if(en_out)
    {
        saida += system("mkdir -p output/images");
	saida += system("rm output/images/*.blob -f");
    }
    if(saida)
    {
    	printf("Erro limpando diretórios!");
    }

    //zera matrizes
    hipMemset(Px, 0, X*Z*2*sizeof(float));
    hipMemset(Pz, 0, X*Z*2*sizeof(float));
    hipMemset(Ax, 0, X*Z*2*sizeof(float));
    hipMemset(Az, 0, X*Z*2*sizeof(float));
    hipMemset(integral_source, 0, n_source*sizeof(float));

    // copia condicoes iniciais
    hipMemcpy(Px, initial, X*Z*2*sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(Pz, initial, X*Z*2*sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(Ax, initial, X*Z*2*sizeof(float), hipMemcpyDeviceToDevice);
    hipMemcpy(Az, initial, X*Z*2*sizeof(float), hipMemcpyDeviceToDevice);


    // simulacao da propagação
    // primeiro frame é condicao de contorno, logo nao calculado
    for(int t = 1; t<T; t++)
    {
	// Atualiza as pressões
	simulateFrameP<<<blockGrid, threadGrid>>>(Px, Pz, Ax, Az, d_x, d_z, cquad, dx, dz, dt, t, X, Z);
	hipDeviceSynchronize();

	//soma termos de fonte
	somaFonteIntegral<<<n_blocksF, threadGrid>>>(Px, Pz, X, Z, T, t, n_source, pos_source_x, pos_source_z, source, integral_source, 0, idx_source);
	hipDeviceSynchronize();

	// grava resultado nos sensores
	gravaBufferSensores<<<n_blocksS, threadGrid>>>(Px, Pz, record_buffer, X, Z, T, t, n_sensor, pos_sensor_x, pos_sensor_z);
	hipDeviceSynchronize();

	// Atualiza as velocidades
	simulateFrameA<<<blockGrid, threadGrid>>>(Px, Pz, Ax, Az, d_x, d_z, cquad, dx, dz, dt, t, X, Z);
	hipDeviceSynchronize();
		
	
	// gera arquivos de saída
	if(en_out)
	{
	    // soma Px e Pz para exibição
	    somaFrames<<<blockGrid, threadGrid>>>(P, &Px[xzt(0,0,t)], &Pz[xzt(0,0,t)], X, Z);
	    hipMemcpy(frame_buffer, P, X*Z*sizeof(float), hipMemcpyDeviceToHost);
	    writeFramePipe (pipeout, frame_buffer, X, Z, t, pos_sensor_x, pos_sensor_z, n_sensor);
	    //writeFrame (P, X, Z, t, pos_sensor_x, pos_sensor_z, n_sensor);
        }
    }

    
    // copia dados do device pro host
    hipMemcpy (recording_h, record_buffer, T * n_sensor * sizeof (float), hipMemcpyDeviceToHost);

    // fecha pipe do video
    if(en_out)
    {
	fflush(pipeout);
	pclose(pipeout);
	hipFree(frame_buffer);
    }
}



